#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>

#define THREAD_COUNT 1024
// Max device memory : 4 GB
#define MAX_MEMORY ((long long)4e9)


void read_graph(std::string fname, int *&row_ptr, int *&col_ind, int &num_nodes, int &num_edges, bool zero_based = false)
{
    std::ifstream input(fname.c_str());
    if (input.fail())
        throw "No file is found in the current path!";

    // read graph
    std::string line = "%";
    while (line.find("%") != std::string::npos)
    {
        getline(input, line);
    }

    std::istringstream ss(line.c_str());
    ss >> num_nodes >> num_nodes >> num_edges;
    int edge_cnt = 0;
    int v1, v2;
    std::vector< std::vector<int> > adj_list(num_nodes);
    for (int i = 0; i < num_edges; i++)
    {
        getline(input, line);
        std::istringstream inp(line.c_str());
        inp >> v1 >> v2;
        if (!zero_based)
        {
            v1--; // make it 0 based
            v2--;
        }
        if (v1 != v2)
        {
            adj_list[v1].push_back(v2); // add the edge v1->v2
            adj_list[v2].push_back(v1); // add the edge v2->v1
            edge_cnt++;
        }
    }
    input.close();
    num_edges = edge_cnt;

    hipHostMalloc((void **)&row_ptr, sizeof(int) * (num_nodes + 1), hipHostMallocDefault);
    hipHostMalloc((void **)&col_ind, sizeof(int) * (2 * num_edges), hipHostMallocDefault);

    row_ptr[0] = 0;
    int index = 0;
    for (int v = 0; v < num_nodes; v++)
    {
        row_ptr[v + 1] = adj_list[v].size(); // assign number of edges going from node v
        for (int i = 0; i < (int)adj_list[v].size(); i++)
        {
            col_ind[index] = adj_list[v][i]; // put all edges in order wrt row_ptr
            index++;
        }
    }
    for (int v = 1; v < num_nodes + 1; v++)
    { // cumulative sum
        row_ptr[v] += row_ptr[v - 1];
    }
}

__global__
void cent_kernel(float *results, int *dist, int *sigma, float *delta, int *rp, int *ci, int n) {
    
    __shared__ int level;
    __shared__ int improved;
    for(int s = blockIdx.x; s < n; s += gridDim.x) {
        if(threadIdx.x == 0) {
	        results[s] = rp[s + 1] - rp[s]; // degree 1
	        level = 0;
	        improved = 1;
                dist[s * n + s] = 0;
                sigma[s * n + s] = 1;
	    }
	    __syncthreads();
        
	    // BFS
        while(improved == 1) {
            if(threadIdx.x == 0) improved = 0;
            for(int node = threadIdx.x; node < n; node += blockDim.x) {
                for(int edge = rp[node]; edge < rp[node + 1]; edge++) {
                    int adj = ci[edge];
                    if(dist[(s * n) + adj] == level && dist[(s * n) + node] == -1) {
                        dist[(s * n) + node] = level + 1;
                        improved = 1;
                    }
                    if(dist[(s * n) + adj] == level && dist[(s * n) + node] == level + 1) {
                        sigma[(s * n) + node] += (float) sigma[(s * n) + adj];
                    }
                }
            }
            if(threadIdx.x == 0) level++;
            __syncthreads();
        }

        int dist_sum = 0;
        int dist2_cnt = 0;

        // DISTANCE ADDER
        if(threadIdx.x == 0) {
            for(int i = 0; i < n; i++) {
                if(dist[(s * n) + i] > 0) {
                    if(dist[(s * n) + i] <= 2) dist2_cnt++;
                    dist_sum += dist[(s * n) + i];
                }
            }
            results[n + s] = dist2_cnt; // degree 2
            results[2 * n + s] = (float) n / dist_sum; // closeness cent.
        }

	    while(level > 0) {
	        for(int node = threadIdx.x; node < n; node += blockDim.x) {
                if(dist[s * n + node] == level){
                    for(int edge = rp[node]; edge < rp[node + 1]; edge++) {
                        int adj = ci[edge];
                        if(dist[(s * n) + adj] + 1 == dist[(s * n) + node]) {
                            atomicAdd(&delta[(s * n) + adj], (sigma[(s * n) + adj] * 1.0) / sigma[(s * n) + node] * (1 + delta[(s * n) + node]));
                        }
                    }
                    atomicAdd(&results[3 * n + node], delta[(s * n) + node] / 2);
                }
            }
            if(threadIdx.x == 0) level--;
            __syncthreads();
	    }
    }
}

float* compute_centralities(int *rp, int *ci, int n, float &time_taken) {
    const int BLOCK_COUNT = MAX_MEMORY / (4 * 3 * n);
    int *sigma, *dist;
    float *delta, *d_results;

    hipMalloc((void **)&d_results, sizeof(float) * n * 4);
    hipMalloc((void **)&sigma, sizeof(int) * n * BLOCK_COUNT);
    hipMalloc((void **)&dist, sizeof(int) * n * BLOCK_COUNT);
    hipMalloc((void **)&delta, sizeof(float) * n * BLOCK_COUNT);

    hipMemset(dist, -1, sizeof(int) * n * BLOCK_COUNT);
    hipMemset(sigma, 0, sizeof(int) * n * BLOCK_COUNT);
    hipMemset(delta, 0, sizeof(float) * n * BLOCK_COUNT);
    hipMemset(d_results, 0, sizeof(float) * 4 * n);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    cent_kernel<<<BLOCK_COUNT, THREAD_COUNT>>>(d_results, dist, sigma, delta, rp, ci, n);
    hipDeviceSynchronize();
    
    hipEventRecord(end);
    hipEventSynchronize(end);

    hipEventElapsedTime(&time_taken, start, end);

    float *results;
    hipHostMalloc((void **)&results, sizeof(float) * n * 4, hipHostMallocDefault);
    hipMemcpy(results, d_results, sizeof(float) * n * 4, hipMemcpyDeviceToHost);

    hipFree(sigma);
    hipFree(dist);
    hipFree(delta);
    hipFree(d_results);
    
    hipDeviceSynchronize();
    return results;
}

int main()
{
    hipSetDevice(0);

    std::string filename = "../data/wing_nodal.mtx";
    int *row_ptr, *col_ind;
    int num_nodes, num_edges;
    read_graph(filename, row_ptr, col_ind, num_nodes, num_edges);
    printf("[INFO] Graph is read: %s.\n", filename.c_str());

    int *rp;
    int *ci;

    hipMalloc((void **)&rp, sizeof(int) * (num_nodes + 1));
    hipMalloc((void **)&ci, sizeof(int) * (num_edges * 2));
    printf("[INFO] CUDA memory parameters are allocated for kernel function.\n");

    hipMemcpy(rp, row_ptr, sizeof(int) * (num_nodes + 1), hipMemcpyHostToDevice);
    hipMemcpy(ci, col_ind, sizeof(int) * (num_edges * 2), hipMemcpyHostToDevice);

    printf("[INFO] CUDA memory parameters are set for kernel function.\n");
  
    float time_taken;  
    float *results = compute_centralities(rp, ci, num_nodes, time_taken);

    printf("[INFO] Kernel function is finished.\n");

    printf("Centrality Results:\n");
    for (int i = 0; i < num_nodes; i++)
    {
        printf("%.5f; %.5f; %.5f; %.5f\n ", results[i], results[num_nodes + i], results[2 * num_nodes + i], results[3 * num_nodes + i]);
    }
    printf("[INFO] Kernel call is ended in: %.5f ms.\n", time_taken);

    hipHostFree(results);
    hipHostFree(row_ptr);
    hipHostFree(col_ind);
    hipFree(rp);
    hipFree(ci);

    return 0;
}


