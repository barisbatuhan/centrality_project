/*
 @author furkanergun
 @author barisbatuhan
 @author halpar
 @author caghankoksal
                        @sabanciuniv.edu
 */
#include <iostream>
#include <hip/hip_runtime.h>
#ifndef UGRAPH_H
#define UGRAPH_H

#include <vector>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <string>
#include <dirent.h>
#include <cmath>
#include <omp.h>
#include <iostream>

class UGraph
{
public:
    
    std::vector<int> row_ptr;
    std::vector<int> col_ind;
    int num_nodes;
    int num_edges;
    int * rowPtr;
    int * colPtr;
    // getter
    int getNumNodes(){
        return num_nodes;
    }
    
    int getNumEdges(){
        return num_edges;
    }
    int * getRowPtr(){
        //allocates space
        int * rowPtr = new int[row_ptr.size()];
        int * itr = rowPtr;
        for(auto &i : row_ptr){
            *itr=i;
            itr++;
        }
        return rowPtr;
    }
    int * getColPtr(){
        //allocates space
        int * colPtr = new int[col_ind.size()];
        int * itr = colPtr;
        for(auto &i : col_ind){
            *itr=i;
            colPtr++;
        }
        return colPtr;
    }
    // constructor
    UGraph(std::string fname);
    UGraph(int node_cnt, int edge_cnt); // random graph generation

    // centrality function
    void compute_centralities(std::vector<std::vector<float>> &result, const std::vector<bool> &requested);
    
    // centrality helper functions
    int bfs_topdown(std::vector<int> &dist, int source, std::vector<double> &sigma, std::vector<int> &queue,
                    std::vector<int> &dist_counter, const std::vector<bool> &requested, int &queue_size);
    float node_closeness(std::vector<int> &dist_counter, int max_dist);
    void node_betweenness(std::vector<std::vector<float>> &result, std::vector<int> &queue,
                          std::vector<int> &dist, std::vector<double> &sigma, int &queue_size);

    // static methods
    static void get_filenames(std::vector<std::string> &filenames, const std::vector<std::string> &locations);
    static void print_centralities(std::string filename, std::vector<std::vector<float>> &result);

private:
    std::string family;
    std::string relative_path;
};

/* CONSTRUCTOR METHODS */

UGraph::UGraph(std::string fname)
{
    std::ifstream input(fname.c_str());
    if (input.fail())
    {
        throw "No file is found in the current path!";
    }
    else
    {
        relative_path = fname;
    }
    // read Ugraph
    std::string line = "%";
    family = "%";
    while (line.find("%") != std::string::npos)
    {
        getline(input, line);
        if (family == "%" && line.find("kind:") != std::string::npos)
        {
            family = line.substr(8);
            family = family.substr(0, family.length() - 1);
        }
    }

    std::istringstream ss(line);
    ss >> num_nodes >> num_nodes >> num_edges;
    int v1, v2;
    double weight;

    std::vector<std::vector<int>> adj_list(num_nodes);
    for (int i = 0; i < num_edges; i++)
    {
        getline(input, line);
        std::istringstream inp(line);
        inp >> v1 >> v2;
        v1--; // make it 0 based
        v2--;

        if (v1 != v2)
        {
            adj_list[v1].push_back(v2); // add the edge v1->v2
            adj_list[v2].push_back(v1); // add the edge v2->v1
        }
    }

    row_ptr = std::vector<int>(num_nodes + 1);
    col_ind = std::vector<int>(2 * num_edges);
    row_ptr[0] = 0;
    int index = 0;
    for (int v = 0; v < num_nodes; v++)
    {
        row_ptr[v + 1] = adj_list[v].size(); // assign number of edges going from node v
        for (int i = 0; i < (int)adj_list[v].size(); i++)
        {
            col_ind[index] = adj_list[v][i]; // put all edges in order wrt row_ptr
            index++;
        }
    }
    for (int v = 1; v < num_nodes + 1; v++)
    { // cumulative sum
        row_ptr[v] += row_ptr[v - 1];
    }
}

UGraph::UGraph(int node_cnt, int edge_cnt)
{
    srand(112);
    std::vector<std::vector<bool>> adj_list(node_cnt, std::vector<bool>(node_cnt, false));
    num_nodes = node_cnt;
    num_edges = edge_cnt;
    int edge_num = edge_cnt;
    while (edge_num > 0)
    {
        int node1 = rand() % node_cnt;
        int node2 = rand() % node_cnt;
        if (node1 == node2)
            continue;
        if (adj_list[node1][node2] == false)
        {
            adj_list[node1][node2] = true;
            adj_list[node2][node1] = true;
            edge_num--;
        }
    }
    row_ptr = std::vector<int>(node_cnt + 1);
    col_ind = std::vector<int>(2 * edge_cnt);
    row_ptr[0] = 0;
    int index = 0;
    for (int v = 0; v < node_cnt; v++)
    {
        int adj_cnt = 0;
        for (int i = 0; i < (int)adj_list[v].size(); i++)
        {
            if (adj_list[v][i] == true)
            {
                col_ind[index] = i; // put all edges in order wrt row_ptr
                index++;
                adj_cnt++;
            }
        }
        row_ptr[v + 1] = row_ptr[v] + adj_cnt; // assign number of edges going from node v
    }
}

/* CENTRALITY METHODS */

int UGraph::bfs_topdown(std::vector<int> &dist, int source, std::vector<double> &sigma, std::vector<int> &queue,
                        std::vector<int> &dist_counter, const std::vector<bool> &requested, int &queue_size)
{
    queue[0] = source;
    dist[source] = 0;
    sigma[source] = (double) 1 / (num_nodes * num_nodes);
    dist_counter[0]++;
    int front = 0, max_distance = 0;
    queue_size = 1;

    while(front < queue_size) {
        int &v = queue[front];
        if(dist[v] == 2 && !requested[2] && !requested[3]) break;
        front++;
        for(int edge = row_ptr[v]; edge < row_ptr[v + 1]; edge++) {
            int &w = col_ind[edge];
            if(dist[w] < 0) { // w not visited yet
                queue[queue_size] = w;
                queue_size++;
                dist[w] = dist[v] + 1;
                if(max_distance < dist[w]) max_distance = dist[w];
                dist_counter[dist[w]]++;
            }
            if(requested[3] && dist[w] == dist[v] + 1) {
                sigma[w] += sigma[v];
            }
        }
    }

    return max_distance;
}

float UGraph::node_closeness(std::vector<int> &dist_counter, int max_dist)
{
    int sum = 0;
    for(int i = 1; i <= max_dist; i++) {
        sum += i * dist_counter[i];
    }
    return (float) 1 / sum;
}

void UGraph::node_betweenness(std::vector<std::vector<float>> &result, std::vector<int> &queue,
                              std::vector<int> &dist, std::vector<double> &sigma, int &queue_size)
{
    std::vector<float> delta(num_nodes, 0);
    for(int index = queue_size - 1; index > 0; index--) {
        int &w = queue[index];
        if(w == -1) continue;
        for(int edge = row_ptr[w]; edge < row_ptr[w + 1]; edge++) {
            int &v = col_ind[edge];
            if(dist[v] + 1 == dist[w]) { // v is parent of w
                delta[v] += (float) (sigma[v] / sigma[w]) * (1 + delta[w]);
            }
        }
        #pragma omp atomic
        result[3][w] += delta[w];
    }
    return;
}

void UGraph::compute_centralities(std::vector<std::vector<float>> &result, const std::vector<bool> &requested)
{
    result = std::vector<std::vector<float>>(4, std::vector<float>(num_nodes, 0));

    #pragma omp parallel for shared(result) schedule(dynamic)
    for(int s = 0; s < num_nodes; s++) { // for all source vertices
        // Parameter Initialization
        std::vector<int> dist(num_nodes, -1);            // distance array from source to vertices
        std::vector<int> dist_counter(num_nodes, 0);     // counts nodes for each distances
        std::vector<double> sigma(num_nodes, 0);            // number of shortest paths from source to index
        std::vector<int> queue(num_nodes, -1);           // non-decreasing read order for bfs
        int queue_size = 0;
        
        // Degree 1 calculation
        if(requested[0]) result[0][s] = (float) row_ptr[s + 1] - row_ptr[s];
        // BFS - maximum distance (diameter of graph) is returned
        int max_dist = bfs_topdown(dist, s, sigma, queue, dist_counter, requested, queue_size);
        // Degree 2 calculation
        if(requested[1]) result[1][s] = (float) dist_counter[2];
        // Closeness calculation
        if(requested[2]) result[2][s] = node_closeness(dist_counter, max_dist);
        // Betweenness calculation
        if(requested[3]) node_betweenness(result, queue, dist, sigma, queue_size);
    }
}

/* STATIC METHODS */
void UGraph::get_filenames(std::vector<std::string> &filenames, const std::vector<std::string> &locations)
{
    for (int i = 0; i < locations.size(); i++)
    {
        if (auto dir = opendir(locations[i].c_str()))
        {
            while (auto f = readdir(dir))
            {
                if (!f->d_name || f->d_name[0] == '.')
                    continue;

                std::string path = locations[i] + f->d_name;
                filenames.push_back(path);
            }
        }
    }
}

void UGraph::print_centralities(std::string filename, std::vector<std::vector<float>> &result) {
    std::cout << std::endl
         << "--------------------------------------------------------------------" << std::endl
         << "| Results for " << filename << std::endl
         << "--------------------------------------------------------------------" << std::endl
         << "|   Indx     |    Deg1    |    Deg2    |    Clos    |    Betw    |" << std::endl
         << "   ------        ------       ------       ------       ------   " << std::endl;

    for(int i = 0; i < result[0].size(); i++) {
        std::cout << "|";
        std::string index = std::to_string((float) i);
        if(i >= 100) std::cout << " " <<  index << " |";
        else if(i >= 10) std::cout << " " <<  index << "  |";
        else if(i < 0) std::cout << " " <<  index << "  |";
        else std::cout << "  " <<  index << "  |";
        
        for(int j = 0; j < 4; j++) {
            std::string res = std::to_string(result[j][i]);
            // float res = result[j][i];
            if(result[j][i] >= 100) std::cout << " " <<  res << " |";
            else if(result[j][i] >= 10) std::cout << " " <<  res << "  |";
            else if(result[j][i] < 0) std::cout << " " <<  res << "  |";
            else std::cout << "  " <<  res << "  |";
        }
        std::cout << std::endl;
    }
}

#endif

#define mem_per_block 49152
#define mem_const 65536
#define warp_size 32
//max global memory is 11441mb
#define global_mem 11441e6
#define max_thread_per_block 1024
#define INF 99999
#define DEBUG
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
  }
  #endif
  return result;
}

/*
 
 adjecancyListPointers == row_ptr
 adjecancyList == col_ptr
 */

__global__ void betweenKernel(double * result, double * preq, UGraph * graph, int * sigma, int * dist, int *pre, int *prepointer, int no_nodes){
    int tid = threadIdx.x;
    int bDim = blockDim.x;
    if(tid<no_nodes){
        __shared__ int preLen;
        __shared__ int prePtrLen;
        __shared__ int iter;
        if(!tid)
            iter=-1; //init
        __syncthreads();
        while(iter < (no_nodes-1)){
            if(!tid){
                pre[0] = ++iter;
                prePtrLen = 1;
                prePtrLen = 1;
                prepointer[0] = 0;
                prepointer[1] = 1;
            }
            __syncthreads();
            for(int vertex = tid; vertex<no_nodes; vertex+=bDim){
                preq[vertex] = 0.0;
                if(vertex == iter){
                    //same
                    dist[vertex] = 0;
                    sigma[vertex] = (double) 1 / (no_nodes * no_nodes);
                }
                else{
                    dist[vertex] = INF;
                    sigma[vertex] = 0;
                }
            }
            __syncthreads();
            
            bool cont=true;
            while(cont){
                __syncthreads();
                for(int i=tid; i<prepointer[prePtrLen]; i+=bDim){
                    if(i>=prepointer[prePtrLen-1]){
                        int vertex = pre[i]; //no bad access
                        int * rowPtrLoc = graph->rowPtr;
                        for(int j = rowPtrLoc[vertex]; j<rowPtrLoc[vertex+1]; j++){
                            int ed = graph->colPtr[j];
                            int el = atomicCAS(&dist[ed], INF, dist[vertex]+1);
                            if(el == INF){
                                int res = atomicAdd(&preLen, 1);
                                pre[res] = ed;
                            }
                            if((dist[ed]+1) == dist[vertex]){
                                atomicAdd(&sigma[ed], sigma[vertex]);
                            }
                        }
                    }
                }
                __syncthreads();
                if(preLen == prepointer[prePtrLen])
                    cont=false; // no upd
                if(cont && !tid){
                    prePtrLen+=1;
                    prepointer[prePtrLen] = preLen;
                }
                if(cont)
                    __syncthreads();
            }
            
            while(prePtrLen){
                for(int i=tid; i<prepointer[prePtrLen]; i+=bDim){
                    if(i>=prepointer[prePtrLen-1]){
                        int vertex = pre[i];
                        int * rowPtrLoc = graph->rowPtr;
                        for(int j=rowPtrLoc[vertex]; j<rowPtrLoc[vertex+1]; j++){
                            int ed = graph->colPtr[j];
                            if((dist[ed]-1)==dist[vertex])
                                if(sigma[ed])
                                    preq[vertex] += (preq[ed]+1) * (double(sigma[vertex])/sigma[ed]);
                            
                            if(vertex!=iter){
                                result[vertex] += preq[vertex]*(1/2); //adding half
                            }
                                
                        }
                    }
                }
                __syncthreads();
                
                if(!tid)
                    prePtrLen-=1;
                __syncthreads();
            }
        }
    }
}



int main(){
    //host
    UGraph * hostGraph = new UGraph("data/small/epb1.mtx");
    int no_nodes = hostGraph->getNumNodes();
    int no_edges = hostGraph->getNumEdges();
    int * row_ptr = hostGraph->getRowPtr();
    int * col_ptr = hostGraph->getColPtr();
    int rowSize = no_nodes+1;
    int colSize = 2*no_edges+1;
    
    //device
    UGraph * devGraph;
    int devNodes;
    int devEdges;
    int * devRow;
    int * devCol;
    
    
    //centrality host
    double * result = new double[no_nodes];
    
    //centrality device
    int *sigma;
    double *devResult;
    double *preq;
    int *dist;
    int *pre;
    int *prepointer;
    printf("No nodes: %d\n", no_nodes);
    printf("No edges: %d\n", no_edges);
    printf("Started cuda allocations\n");
    hipMalloc((void **)&devGraph, sizeof(UGraph));
    hipMalloc((void **)&row_ptr, sizeof(int) * rowSize);
    hipMalloc((void **)&col_ptr, sizeof(int) * colSize);
    hipMalloc((void **)&devResult , sizeof(double) * no_nodes);
    hipMalloc((void **)&preq , sizeof(double) * no_nodes);
    hipMalloc((void **)&sigma , sizeof(int) * no_nodes);
    hipMalloc((void **)&dist , sizeof(int) * no_nodes);
    hipMalloc((void **)&pre , sizeof(int) * (no_nodes+1)); //this two needs additional memory
    hipMalloc((void **)&prepointer , sizeof(int) * (no_nodes+1));  //this two needs additional memory
    printf("Ended cuda allocations\n");
    printf("Started cuda memory copying\n");
    hipMemcpy(devGraph, hostGraph, sizeof(UGraph), hipMemcpyHostToDevice);
    hipMemcpy(devRow, row_ptr, sizeof(int) * rowSize, hipMemcpyHostToDevice);
    hipMemcpy(&(devGraph->rowPtr), &devRow, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(devCol, col_ptr, sizeof(int) * colSize, hipMemcpyHostToDevice);
    hipMemcpy(&(devGraph->colPtr), &devCol, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(devResult, result, sizeof(double) * no_nodes, hipMemcpyHostToDevice);
    printf("Ended cuda memory copying\n");
    printf("Starting kernel\n");
    betweenKernel<<<1, 256>>>(devResult, preq, devGraph, sigma, dist, pre, prepointer, no_nodes);
    printf("Kernel ended\n");
    printf("Copying back the result to memory\n");
    hipMemcpy(result, devResult, sizeof(double) * no_nodes, hipMemcpyDeviceToHost);
    for(int i=0; i<no_nodes; i++){
        printf("%d: %d\n", i, result[i]);
    }
    printf("Ended");
    hipFree(devGraph);
    hipFree(devRow);
    hipFree(devCol);
    hipFree(sigma);
    hipFree(devResult);
    hipFree(preq);
    hipFree(dist);
    hipFree(pre);
    hipFree(prepointer);
}
